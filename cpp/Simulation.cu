#include "hip/hip_runtime.h"
#include "Simulation.hpp"

using namespace std;
using namespace arma;

#include "Simulation_kernels.cu.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
// Put these after kernel calls
//gpuErrchk( hipPeekAtLastError() );
//gpuErrchk( hipDeviceSynchronize() );


inline numtype cpu_sqrt(numtype x){
	#if NUMTYPE_IS_FLOAT
	return sqrtf(x);
	#else
	return sqrt(x);
	#endif
}

inline numtype cpu_sqr(numtype x){
	return x * x;
}

inline numtype cpu_pow(numtype x, numtype y){
	#if NUMTYPE_IS_FLOAT
	return powf(x, y);
	#else
	return pow(x, y);
	#endif
}

inline numtype cpu_sin(numtype x){
	#if NUMTYPE_IS_FLOAT
	return sinf(x);
	#else
	return sin(x);
	#endif
}

inline numtype cpu_cos(numtype x){
	#if NUMTYPE_IS_FLOAT
	return cosf(x);
	#else
	return cos(x);
	#endif
}

inline numtype cpu_log(numtype x){
	#if NUMTYPE_IS_FLOAT
	return logf(x);
	#else
	return log(x);
	#endif
}

// Constructors /////////////////////////////////////////////////////////////////////////
// Direct constructor
Chains::Chains(int N_max) {

    // Store the maximum number of cells in the simulation
    this->N_max = N_max;

    // Set some default parameters (initialize some default objects)
    P_0                 = 0;        // [1/µm^2] The density of invading phages in the simulation initially

    M_max               = 1e6;      // Maximum number of phages in simulation

    dT                  = 1e-6;     // [hour]   Size of the time step

    nSamp               = 100;      // Number of samples to save per simulation hour
    L                   = -1;       // [µm]     Length of boundary condition box (x direction) (L = -1 sets auto scaling)

    margin              = 1;        // 			Number of target typical length scales to simulate

    Ld                  = 3.00;     // [µm]     The length scale for division (Typical volume 1.33 µm^3)
    R                   = 0.45;     // [µm]     The "radius" of the cells

    k_int               = 500;      // [N * m]    Parameter for internal spring potential
    k_rep               = k_int/2;  // [N * m]    Parameter for repulsive potential
    k_att               = k_int/4;  // [N * m]    Parameter for attraction potential
    k_pull              = 0;        // [N]      Parameter for colony formation (colony gravity)

    gamma               = 1/dT;     //          Probability to infect cell
    beta                = 100;      //          Multiplication factor phage
    delta               = 0.003;    // [1/hour] Rate of phage decay

    r                   = 1/0.5;    // [1/hour] Rate of lysis
    T_i                 = -1;       // [hours]  Time when the phage infections begins (less than 0 disables phage infection)

    eta                 = 0.1;      // Amount of division noise along length axis (width of gaussian)
    nu                  = 0.05;     // Amount of displacement noise of the new poles (width of gaussian)

	bendingAngle        = M_PIl/6;	// [rad]		The allowed angle between two cells

    D_B                 = 0;        // [µm^2/hour] Diffusion constant for the cells
    D_P                 = 13000;    // [µm^2/hour] Diffusion constant for the phage

    Time                = 0.0;      // Counter for how many time steps have passed
    RunTime             = 0.0;  	// Variable tracking total run time

    lockCells           = false; 	// Boolean to lock the cells (stop updating)
    allCaptured         = true;	    // Boolean to stop phage updating (all are captured)

    debug               = 1;        // The amount of information to print to terminal
    exit                = false;    // Boolean to control early exit
    firstRun            = true;     // Bool to indicate if this run is the first (i.e. first time we write data)

    wellMixed           = false;    // Bool to indicate if bacteria should be well mixed

    exportAny           = false;    //
    exportCellData      = false;    // Booleans to control the export output
    exportColonySize    = false;    //
    exportPhageData     = false;    //

    ready               = false;    // Boolean to indicate whether the data is ready on the GPU

    rngSeed = -1;                   // Random number seed  ( set to -1 if unused )

}


// Copy constructor
Chains::Chains(Chains& other) {

    P_0                 = other.P_0;                        // [1/µm^2] The density of invading phages in the simulation initially

    N_max               = other.N_max;                      // Maximum number of cells in simulation
    M_max               = other.M_max;                      // Maximum number of phages in simulation

    dT                  = other.dT;                         // Size of the time step

    nSamp               = other.nSamp;                      // Number of samples to save
    L                   = other.L;                          // [µm]     Length of boundary condition box

    margin              = other.margin;

    Ld                  = other.Ld;                         // [µm]     The length scale for division (Typical volume x.x µm^3)
    R                   = other.R;                          // [µm]     The "radius" of the cells

    k_rep               = other.k_rep;                      // [N * m]    Parameter for repulsive potential
    k_att               = other.k_att;                      // [N * m]    Parameter for attraction potential
    k_int               = other.k_int;                      // [N * m]    Parameter for internal spring potential
    k_pull              = other.k_pull;

    gamma               = other.gamma;                      //          Probability to infect cell
    beta                = other.beta;
    delta               = other.delta;                      // [1/hour] Rate of phage decay

    r                   = other.r;
    T_i                 = other.T_i;                        // [hours]  Time when the phage infections begins (less than 0 disables phage infection)

    eta                 = other.eta;                        // Amount of division noise along length axis (width of gaussian)
    nu                  = other.nu;                         // Amount of displacement noise of the new poles (width of gaussian)

    bendingAngle        = other.bendingAngle;

    D_B                 = other.D_B;                        // [µm^2/hour] Diffusion constant for the cells
    D_P                 = other.D_P;                        // [µm^2/hour] Diffusion constant for the phage

    Time                = other.Time;                       // Counter for how many time steps have passed
    RunTime             = other.RunTime;        			// Variable tracking total run time

    lockCells           = other.lockCells;
    allCaptured         = other.allCaptured;

    debug               = other.debug;                      // The amount of information to print to terminal
    exit                = other.exit;                       // Boolean to control early exit
    firstRun            = other.firstRun;                   // Bool to indicate if this run is the first

    wellMixed           = other.wellMixed;

    exportAny           = other.exportAny;                  //
    exportCellData      = other.exportCellData;             // Booleans to control the export output
    exportColonySize    = other.exportColonySize;           //
    exportPhageData     = other.exportPhageData;            //

    ready               = false;                            // Boolean to indicate whether the data is ready on the GPU
    other.ready         = false;

    rngSeed             = other.rngSeed;                    // The seed for the random number generator

    // Copy random number generator
    rng = other.rng;

    // Copy the configuration arguments
    cellsBlockSize = other.cellsBlockSize;
    cellsGridSize  = other.cellsGridSize;

    phagesBlockSize = other.phagesBlockSize;
    phagesGridSize  = other.phagesGridSize;

    // Copy the device pointers
    d_cells        = other.d_cells;
    d_cells_new    = other.d_cells_new;
    d_phages       = other.d_phages;
    d_active       = other.d_active;

    // Copy GPU data to host
    h_cells        = other.d_cells;
    other.h_cells  = other.d_cells;

    h_phages       = other.d_phages;
    other.h_phages = other.d_phages;

    h_active       = other.d_active;
    other.h_active = other.d_active;

    h_rng_state       = other.d_rng_state;
    other.h_rng_state = other.d_rng_state;

}


// Controls the evaluation of the simulation
int Chains::Run(numtype T) {

    if (T < dT) {
        error("Cannot run simulation for less than dT!");
        return 1;
    }

    if (exit) {
        error("Cannot run: exit flag is set!");
        return 1;
    }

    // Get start time
    time_t  tic;
    time(&tic);

    // Things to run only when simulation is initialized
    if (Time == 0.0) {

        // Initialize the simulation matrices
        Initialize();
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        if (exit) return 1;

    } else {

        // Delete Completed.txt
        string path_s = path + "/Completed.txt";
        struct stat info;
        if (stat(path_s.c_str(), &info) == 0 && S_ISREG(info.st_mode)) unlink(path_s.c_str());
    }

    // Check if data is loaded and ready
    if (not ready) {

        // Ensure GPU data is current
        d_cells     = h_cells;
        d_phages    = h_phages;
        d_active    = h_active;
        d_rng_state = h_rng_state;

    }

    // Check if it is time to spawn phages
    SpawnPhages();

    // Check if export has been enabled, and if so, generate a path
    if (exportAny) path = GeneratePath();

    // Check if we have written data before
    if ((firstRun) and ((f_cells.is_open()) or (f_phages.is_open()) or (f_colonySize.is_open()))) { // This can never be true.
        firstRun = false;
    }

    // Write the reproducible command to log.txt
    if (exportAny) WriteLog();

    // Export the start configuration
    if (firstRun) {

        if ( exportCellData          and (debug > 0)) {cout << "\tExporting Cell Position Data" << endl;}
        if ( exportColonySize        and (debug > 0)) {cout << "\tExporting Colony Size" << endl;}
        if ( exportPhageData         and (debug > 0)) {cout << "\tExporting Phage Position Data" << endl;}

        // Export data
        if (exportAny) ExportData(Time);

    }

    // Run the time evolution

    // Determine the number of time steps between samplings
    int nStepsPerSample = (int)round(1/(nSamp * dT));

    // Determine the number of samples to take
    int nSamplings = nSamp * T;

    // Store the current time
    numtype Time_0 = Time;

    // Loop over samplings
    int t = 0;
    for (int n = 1; n <= nSamplings; n++) {

        // Check for exit flag
        if (exit) { break; }

        // Run time inside samples
        while (n * T / nSamplings - t * dT > dT / 2) {

            // Count steps
            t++;

            // Check for exit flag
            if (exit) { break; }

            // Compute the time
            Time = Time_0 + t * dT;

            // Update remaining phages
            if (not allCaptured) {
                PhageUpdateKernel<<<phagesGridSize, phagesBlockSize>>>(thrust::raw_pointer_cast(&d_phages[0]),
                    thrust::raw_pointer_cast(&d_active[0]),
                    thrust::raw_pointer_cast(&d_cells[0]),
                    cpu_sqrt(2 * D_P * dT),
                    L,
                    R,
                    thrust::raw_pointer_cast(&d_rng_state[0]),
                    h_active.size(),
                    N_max,
                    false);
            }

            // Update cells
            if (not lockCells) {

                CellUpdateKernel<<<cellsGridSize, cellsBlockSize>>>(thrust::raw_pointer_cast(&d_cells[0]),
                    thrust::raw_pointer_cast(&d_cells_new[0]),
                    Ld,
                    R,
                    k_rep,
                    k_att,
                    k_int,
                    k_pull,
                    dT,
                    L,
                    N_max);

                thrust::swap(d_cells, d_cells_new);
            }

            // Spawn phages
            SpawnPhages();
        }

        // Export the data
        if (exportAny) ExportData(Time);

        // Show progress bar
        if ((n > 0) and (debug > 0)) {
            cout << "\t[";
            int pos = 60 * static_cast<float>(n) / static_cast<float>(nSamplings);;
            for (int i = 0; i < 60; ++i) {
                if (i <= pos) cout << ".";
                else cout << " ";
            }
            cout << "] " << "\r";
            cout.flush();
        }
    }

    // Get stop time
    time_t  toc;
    time(&toc);

    // Calculate time difference
    RunTime += difftime(toc, tic);
    float seconds = RunTime;
    float hours   = floor(seconds/3600);
    float minutes = floor(seconds/60);
    minutes -= hours * 60;
    seconds -= minutes * 60 + hours * 3600;

    if (debug > 0) {
        cout << endl;
        cout << "Simulation complete after ";
        if (hours > 0.0)   cout << hours   << " hours and ";
        if (minutes > 0.0) cout << minutes << " minutes and ";
        cout  << seconds << " seconds." << endl;
    }

    std::ofstream f_out;
    f_out.open(GetPath() + "/Completed.txt", fstream::trunc);
    f_out << "\tSimulation complete after ";
    if (hours > 0.0)   f_out << hours   << " hours and ";
    if (minutes > 0.0) f_out << minutes << " minutes and ";
    f_out  << seconds << " seconds." << "\n";
    f_out.flush();
    f_out.close();

    // Write success to log
    if (exit) {
        f_log << ">>Simulation completed with exit flag<<" << endl;
    }

    if (exit) {
        return 1;
    } else {
        return 0;
    }
}


// Initialize the simulation
void Chains::Initialize() {
    deb("Initializing", 1);

    // Set the random number generator seed
    if (rngSeed >= 0.0) {
        rng.seed( rngSeed );
    } else {
        static std::random_device rd;
        rng.seed(rd());
    }

    // Initialize the cells
    deb("- Spawning cells", 1);

    // Compute GPU block and grid size
    cellsBlockSize = 256;
    cellsGridSize = (N_max + cellsBlockSize - 1) / cellsBlockSize;

    // Generate new cell vector
    h_cells.reserve(N_max * 8);

    // Initialize chains
    if (not wellMixed) {

        // Keep track of the angles
        numtype theta;
        numtype phi;

        // And keep track of the center
        numtype center[3] = {0.0, 0.0, 0.0};
        for (int n = 0; n < N_max; n++) {

            // Allocate coordinates
            numtype xP, yP, zP;
            numtype xQ, yQ, zQ;
            numtype xR, yR, zR;
            numtype xS, yS, zS;

            // Generate location 1
            if (n == 0) {
                // First iteration, start at (0, 0, 0)
                xP = 0.0;
                yP = 0.0;
                zP = 0.0;

                // Choose random angle
                theta = bendingAngle * rand(rng) + M_PIl / 4;
                phi   =    2 * M_PIl * rand(rng);

                // Generate location 2 coordinates
                xQ = xP + Ld * sin(theta) * cos(phi);
                yQ = yP + Ld * sin(theta) * sin(phi);
                zQ = zP + Ld * cos(theta);

            } else {

                // Choose random angle (giving the vector r)
                theta =     M_PIl * rand(rng);
                phi   = 2 * M_PIl * rand(rng);

                // Compute r dot PQ
                numtype rdotPQ = cpu_sin(theta) * cpu_cos(phi) * (xQ - xP) + cpu_sin(theta) * cpu_sin(phi) * (yQ - yP) + cpu_cos(theta) * (zQ - zP);
                //                              xr                  xPQ                    yr                   yPQ            zr            zPQ

                // Generate U-vector which is perpendicular to PQ
                numtype xU, yU, zU;
                xU = cpu_sin(theta) * cpu_cos(phi) - rdotPQ * (xQ - xP) / cpu_pow(Ld, 2);
                yU = cpu_sin(theta) * cpu_sin(phi) - rdotPQ * (yQ - yP) / cpu_pow(Ld, 2);
                zU = cpu_cos(theta)                - rdotPQ * (zQ - zP) / cpu_pow(Ld, 2);

                // Normalize U
                numtype normU = cpu_sqrt(xU * xU + yU * yU + zU * zU);
                xU /= normU;
                yU /= normU;
                zU /= normU;

                // Choose a random angle to rotate PQ around U
                theta = bendingAngle * rand(rng);

                // Compute rotation (store in temporarily as vector S)
                numtype ct = cpu_cos(theta);
                numtype st = cpu_sin(theta);

                xS = (ct + xU * xU * (1 - ct))           * (xQ - xP) +      (xU * yU * (1 - ct) - zU * st) * (yQ - yP) +      (xU * zU * (1 - ct) + yU * st) * (zQ - zP);
                yS = (     xU * yU * (1 - ct) + zU * st) * (xQ - xP) + (ct + yU * yU * (1 - ct))           * (yQ - yP) +      (yU * zU * (1 - ct) - xU * st) * (zQ - zP);
                zS = (     xU * zU * (1 - ct) - yU * st) * (xQ - xP) +      (yU * zU * (1 - ct) + xU * st) * (yQ - yP) + (ct + zU * zU * (1 - ct))           * (zQ - zP);

                // Move vector R, a distance of two cell radii along the average angle
                xR = xQ + (xQ - xP + xS ) * R / Ld;
                yR = yQ + (yQ - yP + yS ) * R / Ld;
                zR = zQ + (zQ - zP + zS ) * R / Ld;

                // Generate the location of the second pole
                xS += xR;
                yS += yR;
                zS += zR;

                // Store coordinates in xP and xQ
                xP = xR;
                yP = yR;
                zP = zR;

                xQ = xS;
                yQ = yS;
                zQ = zS;
            }

            // Compute center location
            numtype x = 0.5 * (xP + xQ);
            numtype y = 0.5 * (yP + yQ);
            numtype z = 0.5 * (zP + zQ);

            // Store the center
            center[0] += x / N_max;
            center[1] += y / N_max;
            center[2] += z / N_max;

            // Add cell to system
            h_cells.push_back(xP);
            h_cells.push_back(yP);
            h_cells.push_back(zP);
            h_cells.push_back(xQ);
            h_cells.push_back(yQ);
            h_cells.push_back(zQ);

            // Connect the cell to the next cell
            h_cells.push_back(0);
            h_cells.push_back(0);

            // Connect the cells
            if (n > 0) {
                // Connect the cell to the previous cell
                h_cells[8*n+6] = -n;

                // Connect the previous cell to the cell
                h_cells[8*(n-1)+7] = n + 1;
            }

        }

        // Center the chain on (0, 0, 0)
        for (int n = 0; n < N_max; n++) {

            h_cells[8*n+0] -= center[0];
            h_cells[8*n+1] -= center[1];
            h_cells[8*n+2] -= center[2];
            h_cells[8*n+3] -= center[0];
            h_cells[8*n+4] -= center[1];
            h_cells[8*n+5] -= center[2];

        }

        // Update L values (Include safety margin)
        if (L == -1) {

            // Determine radius of smallest sphere that can encapsulate the sphere
            numtype m = 0;

            // Loop over cells
            for (int n = 0; n < N_max; n++) {
                numtype xP = h_cells[8*n+0];
                numtype yP = h_cells[8*n+1];
                numtype zP = h_cells[8*n+2];
                numtype xQ = h_cells[8*n+3];
                numtype yQ = h_cells[8*n+4];
                numtype zQ = h_cells[8*n+5];

                // Compute reach of P and Q coordinates
                numtype rP = xP * xP + yP * yP + zP * zP;
                numtype rQ = xQ * xQ + yQ * yQ + zQ * zQ;

                // Store largest
                if (rP > m) m = rP;
                if (rQ > m) m = rQ;
            }

            // Convert to radius
            m = cpu_sqrt(m)+R;

            // Set new L value
            L = 2 * m * margin;
        }

        // Center the chain on (L, L, L) / 2
        for (int n = 0; n < N_max; n++) {
            for (int j = 0; j < 6; j++) {
                h_cells[8*n+j] += L / 2;
            }
        }

    } else {    // Initialize well mixed

        // Declare variables
        numtype xP, yP, zP;
        numtype xQ, yQ, zQ;
        numtype theta;
        numtype phi;

        // Spawn bacteria
        for (int n = 0; n < N_max; n++) {

            // Draw random location for cell
            xP = -1;
            yP = -1;
            zP = -1;
            xQ = -1;
            yQ = -1;
            zQ = -1;

            // While any are out of bounds
            while ((xP < 0) or (xP > L) or (yP < 0) or (yP > L) or (zP < 0) or (zP > L) or (xQ < 0) or (xQ > L) or (yQ < 0) or (yQ > L) or (zQ < 0) or (zQ > L)) {

                // Generate coordinates
                xP = rand(rng) * L;
                yP = rand(rng) * L;
                zP = rand(rng) * L;

                 // Choose random angle
                theta =     M_PIl * rand(rng);
                phi   = 2 * M_PIl * rand(rng);

                // Generate location 2 coordinates
                xQ = xP + Ld * sin(theta) * cos(phi);
                yQ = yP + Ld * sin(theta) * sin(phi);
                zQ = zP + Ld * cos(theta);

            }

            // Add cell to system
            h_cells.push_back(xP);
            h_cells.push_back(yP);
            h_cells.push_back(zP);
            h_cells.push_back(xQ);
            h_cells.push_back(yQ);
            h_cells.push_back(zQ);

            // Connect the cell to the previous cell
            h_cells.push_back(0);
            h_cells.push_back(0);
        }
    }

    // Copy cell data to GPU
    d_cells     = h_cells;
    d_cells_new = d_cells;

    // Compute GPU block and grid size
    cellsBlockSize = 256;
    cellsGridSize = (N_max + cellsBlockSize - 1) / cellsBlockSize;

    // Set the ready flag
    ready = true;

    // Give warnings
    if (k_rep * dT > 0.05) {
        cout << "Time step might not be small enough! (k_rep * dT = " << k_rep * dT << ")" << endl;
    }
    if (k_att * dT > 0.05) {
        cout << "Time step might not be small enough! (k_att * dT = " << k_att * dT << ")" << endl;
    }
    if (k_int * dT > 0.05) {
        cout << "Time step might not be small enough! (k_int * dT = " << k_int * dT << ")" << endl;
    }
}


// Simulation functions /////////////////////////////////////////////////////////////////

// Spawns phages according to spawning rules
void Chains::SpawnPhages() {

    if (Time < T_i) {return;}

    if (P_0 != 0.0) {
        deb("Spawning Phages", 1);

        // Compute the number of phages and allocate space
        int M = 0;
        if (P_0 > 0) {
            M = (int)(round(P_0 * cpu_pow(L, 3) / 1e12));
        } else {
            M = (int)(round(-P_0));
        }

        // Reset P_0
        P_0 = 0.0;

        // Create thrust array to store phage locations in
        h_phages.reserve(M);
        h_active.reserve(M);
	    phagesBlockSize = 256;
        phagesGridSize = (M + phagesBlockSize - 1) / phagesBlockSize;

        // Spawn phages uniformly within the space
        for (int m = 0; m < M; m++) {

            // Allocate coordinates
            numtype x = -1;
            numtype y = -1;
            numtype z = -1;

            // Use hit and miss method to generate uniformly distributed phages
            while ((x < 0) or (x > L) or (y < 0) or (y > L) or (z < 0) or (z > L)) {
                x = rand(rng) * L;
                y = rand(rng) * L;
                z = rand(rng) * L;
            }

            // Spawn the new phage
            h_phages.push_back(x);
            h_phages.push_back(y);
            h_phages.push_back(z);
            h_active.push_back(1);

        }

        // Copy phage data to GPU
        d_phages = h_phages;
        d_active = h_active;

        // Initialize rng on device
        d_rng_state.resize(M);
        initRNG<<<phagesGridSize, phagesBlockSize>>>(thrust::raw_pointer_cast(&d_rng_state[0]), M);

        // Set the capture boolean
        allCaptured = false;

        // Check for failed phage spawning
        if (M == 0) {
            allCaptured = true;
        }

        // Set the ready flag
        ready = true;

        deb("- Done!", 1);
    }
}

// Replace cell I with beta phages
void Chains::LyseCell(int I) {

    deb("Lysing cell", 1);

    if (I >= N_max) {
        error("Cannot lyse cell, I >= N_max");
    }

    // Copy bacteria to Host
    h_cells  = d_cells;

    // Allocate space for phages
    int M = h_phages.size();
    if (M + 3 * beta > h_phages.capacity()) {
        h_phages.reserve(M + 3 * beta);
        h_active.reserve(M + beta);
    }

    // Create thrust array to store phage locations in
    phagesBlockSize = 256;
    phagesGridSize = (M + 3 * beta + phagesBlockSize - 1) / phagesBlockSize;

    // Extract information of cell I
    numtype xP  = h_cells[8 * I + 0];
    numtype yP  = h_cells[8 * I + 1];
    numtype zP  = h_cells[8 * I + 2];

    numtype xQ  = h_cells[8 * I + 3];
    numtype yQ  = h_cells[8 * I + 4];
    numtype zQ  = h_cells[8 * I + 5];

    // Define end points of the cell
    colvectype P = {xP, yP, zP};
    colvectype Q = {xQ, yQ, zQ};

    // Define coordinate system with PQ as Z axis
    mattype PQ = zeros<mattype>(3, 3);
    PQ.col(0) = Q - P;

    mattype q, r;
    qr( q, r, PQ );

    mattype XYZ = join_horiz( q.cols(1, 2), PQ.col(0) / norm(PQ.col(0)));

    // Define phage location vectors
    colvectype T;

    // Spawn new phages (Uniformly on the interior of the bursted cell)
    for (int b = 0; b < beta; b++) {

        // Generate segment of cell where phage is spawned
        numtype r = rand(rng);

        // Generate a distance
        numtype d = R * rand(rng);

        // Generate location for phage
        numtype theta = M_PIl / 2 * rand(rng);
        numtype phi   = M_PIl * 2 * rand(rng);

        // Determine where on the cell the phage is located
        if (r < (2 * R / 3) / (Ld + 4 * R / 3)) {          // Phage is located in the top half-sphere

            // Determine translation vectors
            T = P;
            T += cos(theta) * cos(phi) * d * XYZ.col(0) + cos(theta) * sin(phi) * d * XYZ.col(1) - sin(theta) * d * XYZ.col(2);


        } else if (r < (4 * R / 3) / (Ld + 4 * R / 3)) {   // Phage is located in the bottom half-sphere

            // Determine translation vector
            T = Q;
            T += cos(theta) * cos(phi) * d * XYZ.col(0) + cos(theta) * sin(phi) * d * XYZ.col(1) + sin(theta) * d * XYZ.col(2);


        } else {                                            // Phage is located along the cylindrical part

            // Determine translation vector
            T = P + rand(rng) * (Q-P);
            T += cos(phi) * d * XYZ.col(0) + sin(phi) * d * XYZ.col(1);

        }

        // Add new phage
        h_phages.push_back(T(0));
        h_phages.push_back(T(1));
        h_phages.push_back(T(2));
        h_active.push_back(1);

    }

    // Set the capture boolean
    if (beta > 0) allCaptured = false;

    // Initialize rng on device
    d_rng_state.reserve(M + 3 * beta);
    initRNG<<<phagesGridSize, phagesBlockSize>>>(thrust::raw_pointer_cast(&d_rng_state[0]), M + 3 * beta);

    // Remove cell I
    d_cells.erase(d_cells.begin() + 8 * I, d_cells.begin() + 8 * (I + 1));
    N_max--;

    // Copy to the GPU
    d_phages = h_phages;
    d_active = h_active;

    // Set the ready flag
    ready = true;

    deb("- Done!", 1);
}

// Autoscale the simulation space
void Chains::AutoScale() {

    // Store P_0 and T_i value
    numtype P_0 = this->P_0;
    numtype T_i = this->T_i;

    // Copy cells to CPU
    h_cells = d_cells;

    // Set L to be smallest posable box
    numtype L_s = 0;

    // If simulating a chain, use small test box
    if (not wellMixed) {

        // Determine most extreme coordinate
        for (int n = 0; n < N_max; n++) {
            for (int j = 0; j < 6; j++) {
                if (abs(h_cells[8 * n + j] - L/2) > L_s) L_s = abs(h_cells[8 * n + j] - L / 2);
            }
        }

        // Add R to extent
        L_s += R;

        // Use the smallest L value
        numtype L_test = max(20.0, 6 * L_s);

        // Center the chain on (L, L, L)
        for (int n = 0; n < N_max; n++) {
            for (int j = 0; j < 6; j++) {
                h_cells[8 * n + j] += (L_test - L) / 2;
            }
        }

        // Update L values
        L = L_test;

    } else {

        // Set L_s value when well-mixed
        L_s = L / 4.0;
    }

    // Copy cells to GPU
    d_cells = h_cells;


    // Auto scale with P_0 = -1e5 first, then -1e6, and then -1e7 if it fails /////////////////////
    numtype M;
    numtype T;
    for (int i = 0; i < 3; i ++) {

        // Overwrite P_0 and T_i value
        if (i == 1)      this->P_0 = P_0 * 10;
        else if (i == 2) this->P_0 = P_0 * 100;
        this->T_i = 0;

        // Spawn phages
        SpawnPhages();

        // Count the number of phages
        int nPhages = h_phages.size() / 3;

        // Take steps until enough phage are adsorbed
        int n = 0;                     // Number of iterations run through
        int t_step = round(1e-2/dT);   // Time between checks

        T = 0.0;        // Elapsed simulation time
        M = 1.0;        // Fraction of free phages (relative to start)

        while ((T < 0.1) and (M > 0.3679)) {
            for (int t = 0; t < t_step; t++) {

                // Increment counter
                n++;

                // Update remaining phages
                PhageUpdateKernel<<<phagesGridSize, phagesBlockSize>>>(thrust::raw_pointer_cast(&d_phages[0]),
                    thrust::raw_pointer_cast(&d_active[0]),
                    thrust::raw_pointer_cast(&d_cells[0]),
                    cpu_sqrt(2 * D_P * dT),
                    L,
                    R,
                    thrust::raw_pointer_cast(&d_rng_state[0]),
                    nPhages,
                    N_max,
                    false);
            }

            // Compute the time
            T = n * dT;

            // Update M
            M = static_cast<numtype>(thrust::reduce(d_active.begin(), d_active.end())) / static_cast<numtype>(h_active.size());

        }

        // Make sure some phages have been adsorbed
        if (M == 1.0) {
            if (i == 0) {
                warning("No hits during autoscaling! Retrying...");

                // Clean up
                h_phages.clear();
                d_phages.clear();

            } else {
                error("No hits during autoscaling!");
            }

        } else {
            break;
        }
    }

    // Estimate adsorption rate eta
    numtype eta = - cpu_pow(L, 3) * cpu_log(M) / T;

    // Set new volume size
    numtype V = - eta / cpu_log(0.9); // Scale so that 90% of phage remain free

    // Scale lengths
    numtype L_new;

    // Set new size
    L_new = cpu_pow(V, 1.0/3.0);
    if (L_new < 2 * 2 * L_s) {

        // If margin is too small, keep the same density of test phages
        P_0 = - P_0 * 1e12 / cpu_pow(L_new, 3.0);

        // Scale the space to be larger
        L_new = 2 * 2 * L_s;

    }

    // If simulating a chain, re-center it
    if (not wellMixed) {

        // Center the chain on (L_new, L_new, L_new) / 2
        for (int n = 0; n < N_max; n++) {
            for (int j = 0; j < 6; j++) {
                h_cells[8 * n + j] += (L_new - L) / 2;
            }
        }

    } else {

        // Redraw well mixed bacteria
        h_cells.clear();

        // Declare variables
        numtype xP, yP, zP;
        numtype xQ, yQ, zQ;
        numtype theta;
        numtype phi;

        // Spawn bacteria
        for (int n = 0; n < N_max; n++) {

            // Allocate coordinates
            xP = -1;
            yP = -1;
            zP = -1;
            xQ = -1;
            yQ = -1;
            zQ = -1;

            // While any are out of bounds
            while ((xP < 0) or (xP > L_new) or (yP < 0) or (yP > L_new) or (zP < 0) or (zP > L_new) or (xQ < 0) or (xQ > L_new) or (yQ < 0) or (yQ > L_new) or (zQ < 0) or (zQ > L_new)) {

                // Generate coordinates
                xP = rand(rng) * L_new;
                yP = rand(rng) * L_new;
                zP = rand(rng) * L_new;

                // Choose random angle
                theta =     M_PIl * rand(rng);
                phi   = 2 * M_PIl * rand(rng);

                // Generate location 2 coordinates
                xQ = xP + Ld * sin(theta) * cos(phi);
                yQ = yP + Ld * sin(theta) * sin(phi);
                zQ = zP + Ld * cos(theta);

            }

            // Add cell to system
            h_cells.push_back(xP);
            h_cells.push_back(yP);
            h_cells.push_back(zP);
            h_cells.push_back(xQ);
            h_cells.push_back(yQ);
            h_cells.push_back(zQ);
            h_cells.push_back(0);
            h_cells.push_back(0);
        }
    }

    // Store new L values
    L = L_new;

    // Update GPU values
    d_cells = h_cells;

    // Clean up
    h_phages.clear();
    d_phages.clear();

    h_active.clear();
    d_active.clear();

    allCaptured = true;

    // Reset P_0 and T_i value
    this->P_0 = P_0;
    this->T_i = T_i;

    // Update log.txt
    if (exportAny) WriteLog();
}


// Auto relaxes the bacteria
void Chains::Relax() {

    // Relax for 0.1 hour
    Run(0.1);

    // If no overlap possible, return
    if (N_max < 2) return;

    // Create vector for overlaps
    thrust::device_vector<numtype> d_overlaps;
    d_overlaps.resize(N_max - 1);

    // Detect current overlap
    CellOverlaps<<<cellsGridSize, cellsBlockSize>>>(
        thrust::raw_pointer_cast(&d_cells[0]),
        thrust::raw_pointer_cast(&d_overlaps[0]),
        R,
        N_max);

    numtype maxOverlap = *(thrust::max_element(d_overlaps.begin(), d_overlaps.end()));

    // While overlap exists, advance time
    while (maxOverlap / R > 0.01) {

        // Relax for 0.1 additional hour
        Run(0.1);

        // Detect current overlap
        CellOverlaps<<<cellsGridSize, cellsBlockSize>>>(
            thrust::raw_pointer_cast(&d_cells[0]),
            thrust::raw_pointer_cast(&d_overlaps[0]),
            R,
            N_max);

        // Determine the overlap
        numtype maxOverlap_new = *(thrust::max_element(d_overlaps.begin(), d_overlaps.end()));

        // Detect convergence
        if (abs(maxOverlap - maxOverlap_new) < 1e-8) {
            break;
        }

        // Save overlap
        maxOverlap = maxOverlap_new;

    }
}


// Equilibrates the phage distribution
void Chains::Equilibrate(numtype T) {

    if (T < dT) {
        return;
    }

    if (Time < T_i) {
        error("Cannot equilibrate before phage invasion begins!");
        return;
    }

    if (exit) {
        error("Cannot equilibrate: exit flag is set!");
        return;
    }

    // Things to run only when simulation is initialized
    if (Time == 0.0) {

        // Initialize the simulation matrices
        Initialize();

        if (exit) return;

    } else {

        // Delete Completed.txt
        string path_s = path + "/Completed.txt";
        struct stat info;
        if (stat(path_s.c_str(), &info) == 0 && S_ISREG(info.st_mode)) unlink(path_s.c_str());

    }

    // Check if it is time to spawn phages
    SpawnPhages();

    // Write the reproducible command to log.txt
    if (exportAny) WriteLog();


    // Run the time evolution

    // Determine the number of time steps to run
    int nSteps = (int)round(T / dT);

    // Store the current time
    numtype Time_0 = Time;

    // Loop over steps
    int t = 0;
    while (T - t * dT > dT / 2) {

        // Check for exit flag
        if (exit) { break; }

        // Count steps
        t++;

        // Compute the time
        Time = Time_0 + t * dT;

        // Update remaining phages
        PhageUpdateKernel<<<phagesGridSize, phagesBlockSize>>>(thrust::raw_pointer_cast(&d_phages[0]),
            thrust::raw_pointer_cast(&d_active[0]),
            thrust::raw_pointer_cast(&d_cells[0]),
            cpu_sqrt(2 * D_P * dT),
            L,
            R,
            thrust::raw_pointer_cast(&d_rng_state[0]),
            h_active.size(),
            N_max,
            true);

        // Spawn phages
        SpawnPhages();
    }

}

// Settings /////////////////////////////////////////////////////////////////////////////
// Set the size of the time-step
void Chains::TimeStep(numtype dT) {this->dT = dT;}


// Set the length of the simulation space
void Chains::SetLength(numtype L) {
    this->L = L;
}


// Set the margin (number of length scales to simulate)
void Chains::SetMargin(numtype margin) {this->margin = margin;}


// Sets the time when the phages should start infecting
void Chains::PhageInvasionStartTime(numtype T_i) {this->T_i = T_i;}


// Sets initial density of the phages (1/µm^3)
void Chains::PhageInitialDensity(numtype P_0) {this->P_0 = P_0;}


// Sets the diffusion constant of the phages
void Chains::PhageDiffusionConstant(numtype D_P) { this->D_P = D_P;}


// Sets rate of the infection increasing in stage
void Chains::PhageInfectionRate(numtype r) {this->r = r;}


// Set the decay rate of the phages
void Chains::PhageDecayRate(numtype delta) {this->delta = delta;}


// Set the size of the bursts
void Chains::PhageBurstSize(int beta) {this->beta = beta;}


// Changes the adsorption parameter gamma
void Chains::PhageAdsorptionParameter(numtype gamma) {this->gamma = gamma;}


// Sets the diffusion constant of the bacteria
void Chains::CellDiffusionConstant(numtype D_B) {this->D_B = D_B;}


// Sets the strength of the repulsive potential
void Chains::CellRepulsiveParameter(numtype k_rep) {this->k_rep = k_rep;}


// Sets the strength of the attractive potential
void Chains::CellAttractiveParameter(numtype k_att) {this->k_att = k_att;}


// Sets the strength of the internal potential
void Chains::CellInternalParameter(numtype k_int) {this->k_int = k_int;}


// Sets the (division)length of the bacteria
void Chains::CellLength(numtype Ld) {this->Ld = Ld;}


// Sets the radius of the bacteria
void Chains::CellRadius(numtype R) {this->R = R;}


// Sets the bending angle between cells
void Chains::CellBendingAngle(numtype bendingAngle){this->bendingAngle = bendingAngle;}


// Locks the cells in their current configuration
void Chains::CellLock() {lockCells = true;}


// Sets the bacteria to be well mixed
void Chains::WellMixed() {wellMixed = true;}


// Sets the bacteria to form a spherical colony (with force k_pull)
void Chains::SphericalColony(numtype k_pull) {this->k_pull = k_pull;}


// Sets the maximum number of phages in the simulation
void Chains::MaxPhageCount(int M_max) {this->M_max = M_max;}

// Helping functions ////////////////////////////////////////////////////////////////////

// Sets the seed of the random number generator
void Chains::SetRngSeed(int n) {
    rngSeed = n;
}

// Debug function (prints "input")
void Chains::deb(const std::string& input, int n) {

    if (debug >= n) {
        std::stringstream stream;
        stream << "<<" << input << ">>" << endl;
        cout << stream.str();
        cout.flush();
    }
}

// Error function (prints "input" and set exit = true)
void Chains::error(const std::string& input) {
    if (not f_log.is_open()) f_log.open(path + "/log.txt", fstream::app);
    cerr << "\t>> " << input << " Exiting...<<" << endl;
    f_log << ">> " << input << " Exiting...<<" << endl;
    exit = true;
}

// Warning function (prints "input")
void Chains::warning(const std::string &input) {
    if (not f_log.is_open()) f_log.open(path + "/log.txt", fstream::app);
    cerr << "\t>> " << input << "<<" << endl;
    f_log << ">> " << input << "<<" << endl;
}

// Write a log.txt file
void Chains::WriteLog() {
    deb("Writing log", 1);
    if ((not f_log.is_open()) and (not exit)) {

        // Open the file stream and write the command
        f_log.open(path + "/log.txt", fstream::out);
        deb("- " + path + "/log.txt", 2);

        // Physical parameters
        f_log << "P_0 = "      << P_0      << "\n";        // Density of invading phages
        f_log << "N_max = "    << N_max    << "\n";        // Maximal allowed cells
        f_log << "M_max = "    << M_max    << "\n";        // Maximum allowed phages

        f_log << "T_i = "      << T_i      << "\n";        // Phage infection time
        f_log << "L = "        << L        << "\n";        // Length of boundary condition box
        f_log << "k_rep = "    << k_rep    << "\n";        // Strength of the repulsion potential
        f_log << "k_att = "    << k_att    << "\n";        // Strength of the attraction potential
        f_log << "k_int = "    << k_int    << "\n";        // Strength of the internal spring potential
        f_log << "k_pull = "   << k_pull   << "\n";        // Parameter for colony formation (colony gravity)

        f_log << "R = "        << R        << "\n";        // Critical radius
        f_log << "Ld = "       << Ld       << "\n";        // The length scale for division
        f_log << "D_B = "      << D_B      << "\n";        // Bacteria diffusion
        f_log << "D_P = "      << D_P      << "\n";        // Phage diffusion
        f_log << "gamma = "    << gamma    << "\n";        // Probability to infect cell
        f_log << "beta = "     << beta     << "\n";        // Multiplication factor phage
        f_log << "delta = "    << delta    << "\n";        // Rate of phage decay
        f_log << "r = "        << r        << "\n";        // Rate of lysis

        f_log << "eta = "      << eta      << "\n";        // Amount of division noise
        f_log << "nu = "       << nu       << "\n";        // Amount of displacement noise

        f_log << "theta = "    << bendingAngle << "\n";    // [rad]		The allowed angle between two cells

        // Non physical parameters
        f_log << "dT = "       << dT       << "\n";        // Time step size
        f_log << "nSamp = "    << nSamp    << "\n";        // Number of samples to save per simulation hour
        f_log << "margin = "   << margin   << "\n";        // The number of length scales to simulate
        f_log << "rngSeed = "  << rngSeed  << "\n";        // Random Number Generator seed
        f_log << "wellMixed = "<< wellMixed<< "\n";        // Output the well mixed boolean

        f_log << "debug = "             << debug            << "\n";
        f_log << "exportAny = "         << exportAny        << "\n";
        f_log << "exportCellData = "    << exportCellData   << "\n";
        f_log << "exportColonySize = "  << exportColonySize << "\n";
        f_log << "exportPhageData = "   << exportPhageData  << endl;

        f_log.close(); // By closing it, we allow it to be overwritten by next .Run()
    }
}


// Set debug level to 0
void Chains::Quiet() { debug=0; };


// Set the debug level to n
void Chains::Debug(int n) {debug = n;}


// Set the number of output samples
void Chains::SetSamples(int nSamp) {this->nSamp = nSamp;};


// File outputs /////////////////////////////////////////////////////////////////////////
// Sets boolean for export function
void Chains::ExportCellData()         { exportCellData          = true; exportAny = true; };
void Chains::ExportColonySize()       { exportColonySize        = true; exportAny = true; };
void Chains::ExportPhageData()        { exportPhageData         = true; exportAny = true; };


void Chains::ExportCellDataNow()      { f_ExportCellData(Time); };
void Chains::ExportPhageDataNow()     { f_ExportPhageData(Time); };

// Master function to export the data
void Chains::ExportData(numtype t) {
    deb("Exporting data", 2);

    // Export the data
    if ( exportCellData ) {
        f_ExportCellData(t);
    }
    if ( exportColonySize ) {
        f_ExportColonySize(t);
    }
    if ( exportPhageData ) {
        f_ExportPhageData(t);
    }

    deb("- Done", 2);
}


// Export the position and size of the cells
void Chains::f_ExportCellData(numtype t) {

    // Verify the file stream is open
    string fileName = "CellData";
    OpenFileStream(f_cells, fileName);

    // Copy from GPU
    h_cells = d_cells;

    // Loop over cells in simulation
    for (int i = 0; i < N_max; i++) {

        // Output format:   (T    x    y    z   r   s)
        f_cells << fixed    << setprecision(5);
        f_cells << setw(8)  << t << "\t";
        f_cells << fixed    << setprecision(8);
        f_cells << setw(12) << h_cells[8 * i + 0] << "\t";
        f_cells << setw(12) << h_cells[8 * i + 1] << "\t";
        f_cells << setw(12) << h_cells[8 * i + 2] << "\t";
        f_cells << setw(12) << h_cells[8 * i + 3] << "\t";
        f_cells << setw(12) << h_cells[8 * i + 4] << "\t";
        f_cells << setw(12) << h_cells[8 * i + 5] << "\t";
        f_cells << fixed    << setprecision(0);
        f_cells << setw(12) << h_cells[8 * i + 6] << "\t";
        f_cells << setw(12) << h_cells[8 * i + 7] << endl;
    }
}


// Export the position and size of the phages
void Chains::f_ExportPhageData(numtype t) {

    // Verify the file stream is open
    string fileName = "PhageData";
    OpenFileStream(f_phages, fileName);

    // Copy from GPU
    h_phages = d_phages;

    // Loop over phages in simulation
    for (int i = 0; i < h_phages.size() / 3; i++) {

        // Output format:   T    x    y    z
        f_phages << fixed    << setprecision(5);
        f_phages << setw(8)  << t << "\t";
        f_phages << setw(12) << h_phages[3 * i + 0] << "\t";
        f_phages << setw(12) << h_phages[3 * i + 1] << "\t";
        f_phages << setw(12) << h_phages[3 * i + 2] << endl;
    }
}


// Export the volume of colony and number of cells.
void Chains::f_ExportColonySize(numtype t) {

    // Verify the file stream is open
    string fileName = "ColonySize";
    OpenFileStream(f_colonySize, fileName);

    // Writes the time, total volume, number of cells,
    // number of lytic stage cells, number of lysogenic cells

    //cout << "d_active.size() = " << d_active.size() << endl;
    //cout << "phagesBlockSkize = " << phagesBlockSize << endl;
    //cout << "phagesGridSize = " << phagesGridSize << endl;
    int M = 0;
    if ((not allCaptured) and (h_active.size() > 0)) {
        M = thrust::reduce(d_active.begin(), d_active.end());
        if (M == 0) allCaptured = true; // If all are captured, stop phage updating
    }

    f_colonySize << fixed    << setprecision(5);
    f_colonySize << setw(8)  << t           << "\t";
    f_colonySize << setw(8)  << M           << endl;
}


// Data Handling ////////////////////////////////////////////////////////////////////////
// Open file-stream if not already opened
void Chains::OpenFileStream(ofstream& stream, string& fileName) {

    // Check that if file stream is open.
    if ((not stream.is_open()) and (not exit)) {

        // Debug info
        if (debug > 0) {cout << "\t\tSaving data to file: " << path << "/" << fileName << ".txt" << endl;}

        // Construct path
        string streamPath;
        streamPath = path + "/" + fileName + ".txt";

        // Open the file stream
        if (firstRun) {
            stream.open(streamPath, fstream::trunc);
        } else {
            stream.open(streamPath, fstream::app);
        }

        // Check stream is open
        if ((not exit) and (not stream.is_open())) {
            cerr << "\t>>Could not open filestream \"" << streamPath << "\"! Exiting..<<" << endl;
            f_log <<  ">>Could not open filestream \"" << streamPath << "\"! Exiting..<<" << endl;
            exit = true;
        };

        // If appending to existing file, do not rewrite the meta data
        if (not firstRun) {
            return;
        }
    }
}


// Generates a save path for data-files
string Chains::GeneratePath() {

    // Generate a directory path
    string prefix = "data";    // Data folder name

    // Create the path variable
    string path_s;

    // Create info object
    struct stat info;

    // Check if user has specified numbered folder
    if (path.empty()) {

        // Add the prefix
        path_s += prefix;

        // Check if path exists
        if (not(stat(path_s.c_str(), &info) == 0 && S_ISDIR(info.st_mode))) {
            // Create path if it does not exist
            mkdir(path_s.c_str(), 0700);
        }

        // Loop over folders in date folder, to find current number
        int currentNumerateFolder = 1;
        DIR * dir;
        if ((dir = opendir (path_s.c_str())) != NULL) {
            struct dirent * ent;
            while ((ent = readdir (dir)) != NULL) {
                if (ent->d_type == DT_DIR) {
                    // Skip . or ..
                    if (ent->d_name[0] == '.') {continue;}
                    currentNumerateFolder++;        // Increment folder number
                }
            }
            closedir (dir);
        }

        // Append numerate folder
        path_s += "/";
        path_s += to_string(currentNumerateFolder);

        // Check if path exists
        if (not(stat(path_s.c_str(), &info) == 0 && S_ISDIR(info.st_mode))) {
            // Create path if it does not exist
            mkdir(path_s.c_str(), 0700);
        }

    } else {    // User has specified a path

        // This path maybe more than one layer deep, so attempt to make it recursively
        int len = path.length();

        // Check if prefix has been added
        if ((len < 4) or (not ((path[0] == 'd') and (path[1] == 'a') and (path[2] == 't') and (path[3] == 'a')))) {
            path_s += prefix;
            path_s += '/';
        }

        // Boolean to see name of first folder
        bool firstFolder = true;

        string folder = "";
        for (int i = 0; i < len; i++) {
            folder += path[i]; // Append char to folder name

            // If separator is found or if end of path is reached, construct folder
            if ((path[i] == '/') or (i == len - 1)) {

                // If separator is found, remove it:
                if (path[i] == '/') folder.pop_back();

                // Check if this is the first sub-folder
                if (firstFolder) {
                    firstFolder = false;

                    // Check if first folder contains date format
                    if (not ((folder.length() == 10) and(folder[4] == '-') and (folder[7] == '-'))) {

                        // Check if path exists
                        if (not(stat(path_s.c_str(), &info) == 0 && S_ISDIR(info.st_mode))) {
                            // Create path if it does not exist
                            mkdir(path_s.c_str(), 0700);
                        }
                    }
                }

                // Append folder to path
                path_s += folder;

                // Make folder
                if (not(stat(path_s.c_str(), &info) == 0 && S_ISDIR(info.st_mode)))
                { // Create path if it does not exist
                    mkdir(path_s.c_str(), 0700);
                }

                // Append a separator
                path_s += "/";

                // Reset folder
                folder = "";
            }
        }

        // Remove last separator
        if (path_s.back() == '/') path_s.pop_back();

    }

    // // Generate state folder  TODO: Change such that it works on the GPU
    // string path_ss = path_s + "/state";
    // if (not(stat(path_ss.c_str(), &info) == 0 && S_ISDIR(info.st_mode)))
    // { // Create path if it does not exist
    //     mkdir(path_ss.c_str(), 0700);
    // }

    // Return the generated path
    return path_s;
}


// Sets the folder number (useful when running parallel code)
void Chains::SetFolderNumber(int number) {path = to_string(number);};


// Sets the folder path (useful when running parallel code)
void Chains::SetPath(const std::string& path) {
    exportAny = true;
    this->path = path;
}


// Get properties ///////////////////////////////////////////////////////////////////////
// Returns the save path
std::string Chains::GetPath() {
    return path;
}


// Returns the time
int Chains::GetTime() {
    return Time;
}


// Returns the time-step dT
numtype Chains::GetDeltaT() {
    return dT;
}

// Clean up /////////////////////////////////////////////////////////////////////////////
// Delete the data folder
void Chains::DeleteFolder() {
    DeleteFolderTree(path.c_str());
}


// Delete folders recursively
void Chains::DeleteFolderTree(const char * directory_name) {

    DIR *            dp;
    struct dirent *  ep;
    char            p_buf[512] = {0};


    dp = opendir(directory_name);

    while ((ep = readdir(dp)) != NULL) {
        // Skip self dir "."
        if (strcmp(ep->d_name, ".") == 0 || strcmp(ep->d_name, "..") == 0) continue;

        sprintf(p_buf, "%s/%s", directory_name, ep->d_name);

        // Is the path a folder?
        struct stat s_buf;
        int IsDirectory = -1;
        if (stat(p_buf, &s_buf)){
            IsDirectory = 0;
        } else {
            IsDirectory = S_ISDIR(s_buf.st_mode);
        }

        // If it is a folder, go recursively into
        if (IsDirectory) {
            DeleteFolderTree(p_buf);
        } else {    // Else delete the file
            unlink(p_buf);
        }
    }

    closedir(dp);
    rmdir(directory_name);
}


// Destructor
Chains::~Chains() {

    // Close file-streams
    if (f_cells.is_open()) {
        f_cells.close();
    }
    if (f_colonySize.is_open()) {
        f_colonySize.close();
    }
    if (f_phages.is_open()) {
        f_phages.close();
    }
    if (f_log.is_open()) {
        f_log.close();
    }
}
