#include "hip/hip_runtime.h"
#include "Simulation.hpp"

using namespace std;

int main(int argc, char** argv){

    // Length of the chains
    std::vector<int>     N    = {     1,     3,    10,    32,   100,   316,  1000,  3162};
    std::vector<int>     runs = {     9,     9,     9,     9,     9,     9,     9,     9};

    std::vector<numtype> theta   = {1.0, 0.8, 0.6, 0.4, 0.2};

    // Number of chains to generate for persistence length measurements
    int repeats = 100;

    // Default parameters
    numtype dT_relax = 1e-4;     // Set the time scale during relaxation

    numtype T  = 1;              // Set the length of the experiment
    numtype dT = 1e-7;           // Set the time-step of the experiment

    // Streptobacillus
    numtype R  = 0.45;
    numtype Ld = 1.5;

    int k = 0;
    // Loop over chain lengths
    for (int n = 0; n < N.size(); n++) {
        for (int t = 0; t < theta.size(); t++) {
            for (int r = 0; r < repeats; r++) {

                // Increment counter
                k++;

                // Print the process bar
                cout << "\r[" << k << " / " << theta.size() * N.size() * repeats << "]" << flush;

                // Set the path
                string pathName = "Chain";
                pathName += "/N_" + to_string(N[n]);

                pathName += "/theta_";
                char buffer[80]; // Create a buffer to store the date¨
                snprintf(buffer, sizeof(buffer), "%.3f", theta[t]);
                pathName += string(buffer);
                pathName += "_pi";

                pathName += "/repeat_" + to_string(r);

                // Check if data is already made
                string path_s1 = "data/" + pathName + "/Completed.txt";
                string path_s2 = "data/" + pathName + "/ColonySize.txt";

                // Check if runs exists and is completed
                bool exists = false;
                struct stat info;
                if ((stat(path_s1.c_str(), &info) == 0 && S_ISREG(info.st_mode)) and (stat(path_s2.c_str(), &info) == 0 && S_ISREG(info.st_mode))) { // Base run is completed

                    // Count sub runs that are completed
                    int completed = 0;

                    // Check if lysis runs are completed
                    if ((r == 0) and (N[n] > 1)) {
                        for (int i = 0; i < min(N[n], 5); i++) {
                            string test_path = "data/" + pathName + "/lysis_" + to_string(i) + "/Completed.txt";
                            if (stat(test_path.c_str(), &info) == 0 && S_ISREG(info.st_mode)) {
                                completed++;
                            }
                        }
                    }

                    // Check if any runs need to be completed
                    if ((r == 0) and (N[n] > 1)) {  // Lysis runs still need to be run
                        if (completed == min(N[n], 5)) continue;
                    } else {    // No runs are needed
                        continue;
                    }

                    // Mark the base run as existing
                    exists = true;
                }

                // Load simulation module
                Chains s(N[n]);
                s.SetRngSeed(r + 100 * t + 1000 * (2 * n));
                s.Debug(0);

                // Set the path
                if (not exists) s.SetPath(pathName);

                s.SetSamples(100);

                // Set the cell dimensions
                s.CellLength(Ld);
                s.CellRadius(R);

                // Set the angle between cells
                s.CellBendingAngle(theta[t] * M_PIl);

                // Let the cells relax
                s.TimeStep(dT_relax);
                s.Relax();

                // Check if the run is not a phage run
                if (r >= runs[n]) {
                    s.ExportCellDataNow();
                    continue;
                }

                // Set the data to export
                if (not exists) s.ExportColonySize();

                // Configure the phage invasion
                s.PhageInvasionStartTime(0.0);
                s.PhageInitialDensity(-1e5);

                // Lock the configuration
                s.CellLock();

                // Configure the time step
                s.TimeStep(dT);

                // Autoscale simulation
                s.AutoScale();

                // Run lysis experiment (for first repeat only)
                if ((N[n] > 1) && (r == 0)) {

                    // Select lysis site
                    for (int i = 0; i < min(N[n], 5); i++) {

                        // Create off-spring simulation
                        Chains t(s);

                        // Increase the sampling
                        t.SetSamples(10000);

                        // Configure phage attack
                        t.PhageInitialDensity(0);
                        t.PhageBurstSize(1e4);

                        if (i == 0) {
                            t.LyseCell(0);
                        } else {
                            t.LyseCell(i * (N[n] - 1) / (min(N[n], 5) - 1));
                        }

                        // Change the pathname
                        string path_s = pathName + "/lysis_" + to_string(i);
                        t.SetPath(path_s);

                        // Check if data is already made
                        path_s = "data/" + path_s + "/Completed.txt";

                        // Check if run exists and is completed
                        struct stat info;
                        if (stat(path_s.c_str(), &info) == 0 && S_ISREG(info.st_mode)) continue;

                        // Set the data to export
                        t.ExportColonySize();

                        // Run the experiment
                        t.Run(T);

                        // Store the final configuration
                        t.ExportCellDataNow();
                        t.ExportPhageDataNow();

                    }
                }

                // Skip the run if already completed
                if (exists) continue;

                // Run the Experiment
                s.Run(T);

                // Store the final configuration
                s.ExportCellDataNow();
                s.ExportPhageDataNow();
            }
        }
    }

    cout << "\rDone!                   " << endl;
	return 0;
}
