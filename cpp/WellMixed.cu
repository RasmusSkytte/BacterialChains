#include "hip/hip_runtime.h"
#include "Simulation.hpp"

using namespace std;

int main(int argc, char** argv){

    // Length of the chains
    std::vector<int>     N       = {     1,     3,    10,    32,   100,   316,  1000};
    std::vector<int>     repeats = {     3,     3,     3,     3,     3,     3,     3};

    // Default parameters
    numtype dT_relax = 1e-4;     // Set the time scale during relaxation

    numtype T  = 1;              // Set the length of the experiment
    numtype dT = 1e-7;           // Set the time-step of the experiment

    // Streptobacillus
    numtype R  = 0.45;
    numtype Ld = 1.5;

    int k = 0;
    // Loop over chain lengths
    for (int n = 0; n < N.size(); n++) {
        for (int r = 0; r < repeats[n]; r++) {

            // Increment counter
            k++;

            // Print the process bar
            cout << "\r[" << k << " / " << std::accumulate(repeats.begin(), repeats.end(), 0) << "]" << flush;

            // Set the path
            string pathName = "WellMixed";
            pathName += "/N_" + to_string(N[n]);
            pathName += "/repeat_" + to_string(r);

            // Check if data is already made
            string path_s = "data/" + pathName + "/Completed.txt";

            // Check if run exists and is completed
            bool exists = false;
            struct stat info;
            if (stat(path_s.c_str(), &info) == 0 && S_ISREG(info.st_mode)) {
                if ((r == 0) and (N[n] > 1)) {
                    int completed = 0;
                    for (int i = 0; i < min(N[n], 5); i++) {
                        string test_path = "data/" + pathName + "/lysis_" + to_string(i) + "/Completed.txt";
                        if (stat(test_path.c_str(), &info) == 0 && S_ISREG(info.st_mode)) {
                            completed++;
                        }
                    }
                    if (completed == min(N[n], 5)) continue;
                } else {
                    continue;
                }
                exists = true;
            }

            // Load simulation module
            Chains s(N[n]);
            s.SetRngSeed(r +  800 * (2 * n));
            s.Debug(0);

            // Set the path
            if (not exists) s.SetPath(pathName);

            s.SetSamples(100);

            // Set the cell dimensions
            s.CellLength(Ld);
            s.CellRadius(R);

            // Set the cells to be dispersed
            s.WellMixed();
            s.SetLength(100);

            // Let the cells relax
            s.TimeStep(dT_relax);
            s.Relax();

            // Configure the phage invasion
            s.PhageInvasionStartTime(0.0);
            s.PhageInitialDensity(-1e5);

            // Set the data to export
            if (not exists) s.ExportColonySize();

            // Lock the configuration
            s.CellLock();

            // Configure the time step
            s.TimeStep(dT);

            // Autoscale simulation
            s.AutoScale();

            // Run lysis experiment (for first repeat only)
            if ((N[n] > 1) && (r == 0)) {

                // Select lysis site
                for (int i = 0; i < min(N[n], 5); i++) {

                    // Create off-spring simulation
                    Chains t(s);

                    // Increase the sampling
                    t.SetSamples(10000);

                    // Configure phage attack
                    t.PhageInitialDensity(0);
                    t.PhageBurstSize(1e4);

                    if (i == 0) {
                        t.LyseCell(0);
                    } else {
                        t.LyseCell(i * (N[n] - 1) / (min(N[n], 5) - 1));
                    }

                    // Change the pathname
                    string path_s = pathName + "/lysis_" + to_string(i);
                    t.SetPath(path_s);

                    // Check if data is already made
                    path_s = "data/" + path_s + "/Completed.txt";

                    // Check if run exists and is completed
                    struct stat info;
                    if (stat(path_s.c_str(), &info) == 0 && S_ISREG(info.st_mode)) continue;

                    // Set the data to export
                    t.ExportColonySize();

                    // Run the experiment
                    t.Run(T);

                    // Store the final configuration
                    t.ExportCellDataNow();
                    t.ExportPhageDataNow();

                }
            }

            // Skip the run if already completed
            if (exists) continue;

            // Run the experiment
            s.Run(T);

            // Store the final configuration
            s.ExportCellDataNow();
            s.ExportPhageDataNow();
        }
    }

    cout << "\rDone!                   " << endl;
	return 0;
}
