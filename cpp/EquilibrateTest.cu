#include "Simulation.hpp"

using namespace std;

int main(int argc, char** argv){

    // Length of the chains
    std::vector<int> N = {1, 32};
    int repeats = 3;
    numtype theta = 0.6;

    // Default parameters
    numtype dT_relax = 1e-4;     // Set the time scale during relaxation

    numtype T = 1;               // Set the length of the experiment
    numtype dT = 1e-7;           // Set the time-step of the experiment

    std::vector<numtype> T_Eq = { 0, 1, 2, 3, 4, 5, 6 };

    // Streptobacillus
    numtype R  = 0.45;
    numtype Ld = 1.5;

    int k = 0;
    // Loop over repeats
    for (int n = 0; n < N.size(); n++) {
        for (int r = 0; r < repeats; r++) {

            // Increment counter
            k++;

            // Print the process bar
            cout << "\r[" << k << " / " << T_Eq.size() * N.size() * repeats << "]" << flush;

            // Set the path
            string pathName = "EquilibriumTest";

            // Check if run exists and is completed
            struct stat info;
            char buffer[80]; // Create a buffer
            int completed = 0;
            for (int m = 0; m < T_Eq.size(); m++) {
                snprintf(buffer, sizeof(buffer), "%.0f", T_Eq[m]);
                string test_path = "data/" + pathName + "/T_" + string(buffer) + "/N_" + to_string(N[n]) + "/repeat_" + to_string(r) + "/Completed.txt";
                if (stat(test_path.c_str(), &info) == 0 && S_ISREG(info.st_mode)) {
                    completed++;
                }
            }
            if (completed == T_Eq.size()) continue;


            // Load simulation module
            Chains s(N[n]);
            s.SetRngSeed(r);
            s.Debug(0);

            s.SetSamples(100);

            // Set the cell dimensions
            s.CellLength(Ld);
            s.CellRadius(R);

            // Set the angle between cells
            s.CellBendingAngle(theta * M_PIl);

            // Let the cells relax
            s.TimeStep(dT_relax);
            s.Relax();

            // Configure the phage invasion
            s.PhageInvasionStartTime(0.0);
            s.PhageInitialDensity(-1e5);

            // Lock the configuration
            s.CellLock();

            // Configure the time step
            s.TimeStep(dT);

            // Autoscale simulation
            s.AutoScale();

            // Create off-spring simulation
            Chains t(s);

            // Run without equilibration time
            snprintf(buffer, sizeof(buffer), "%.0f", T_Eq[0]);
            string path = pathName + "/T_" + string(buffer) + "/N_" + to_string(N[n]) + "/repeat_" + to_string(r);

            // Check if run is completed
            string path_s = "data/" + path + "/Completed.txt";

            // Check if run exists and is completed
            if (not (stat(path_s.c_str(), &info) == 0 && S_ISREG(info.st_mode))) {

                // Set the path name
                s.SetPath(path);

                // Set the data to export
                s.ExportColonySize();

                // Run the experiment
                s.Run(T);

                // Store the final configuration
                s.ExportCellDataNow();
                s.ExportPhageDataNow();

            }

            // Loop over equilibration times
            for (int m = 1; m < T_Eq.size(); m++) {

                // Increment counter
                k++;

                // Print the process bar
                cout << "\r[" << k << " / " << T_Eq.size() * N.size() * repeats << "]" << flush;

                // Let the master simulation equilibrate for T time
                t.Equilibrate(T_Eq[m] - T_Eq[m-1]);

                // Make copy
                Chains u(t);

                // Set new path
                snprintf(buffer, sizeof(buffer), "%.0f", T_Eq[m]);
                path = pathName + "/T_" + string(buffer) + "/N_" + to_string(N[n]) + "/repeat_" + to_string(r);

                // Check if run is completed
                string path_s = "data/" + path + "/Completed.txt";

                // Check if run exists and is completed
                if (not (stat(path_s.c_str(), &info) == 0 && S_ISREG(info.st_mode))) {

                    // Set the path name
                    u.SetPath(path);

                    // Set the data to export
                    u.ExportColonySize();

                    // Run the experiment
                    u.Run(T);

                    // Store the final configuration
                    u.ExportCellDataNow();
                    u.ExportPhageDataNow();

                }
            }
        }
    }

    cout << "\rDone!                   " << endl;
	return 0;
}
