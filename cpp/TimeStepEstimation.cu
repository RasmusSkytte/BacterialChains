#include "hip/hip_runtime.h"
#include "Simulation.hpp"

using namespace std;

inline numtype cpu_pow(numtype x, numtype y){
	#if NUMTYPE_IS_FLOAT
	return powf(x,y);
	#else
	return pow(x,y);
	#endif
}

int main(int argc, char** argv){

	// Length of the chains
    int N = 10;
    numtype theta = 0.6*M_PIl;

    // Default parameters
    numtype dT_relax = 1e-4;     // Set the time scale during relaxation

    numtype T = 1;               // Set the length of the experiment
    std::vector<numtype> dT = {cpu_pow(10,-4.00), cpu_pow(10,-4.50), cpu_pow(10,-5.00), cpu_pow(10,-5.50), cpu_pow(10,-6.00), cpu_pow(10,-6.50), cpu_pow(10,-7.00), cpu_pow(10,-7.50)};

    int repeats = 3;

    // Streptobacillus
    numtype R  = 0.45;
    numtype Ld = 1.5;

    int k = 0;
    // Loop over time steps
    for (int t = 0; t < dT.size(); t++ ) {
        for (int r = 0; r < repeats; r++) {

            // Increment counter
            k++;

            // Print the process bar
            cout << "\r[" << k << " / " << dT.size() * repeats << "]" << flush;

            // Set the path
            string pathName = "TimeStepEstimation";

            pathName += "/dT_1e";
            char buffer[80];                                  // Create a buffer to store the date
            snprintf(buffer, sizeof(buffer), "%.2f", log10(dT[t]));
            pathName += string(buffer);

            pathName += "/repeat_";
            pathName += to_string(r);

            // Check if data is already made
            string path_s = "data/"; // Data folder name
            path_s += pathName;
            path_s += "/Completed.txt";

            // Check if run exists and is completed
            struct stat info;
            if (stat(path_s.c_str(), &info) == 0 && S_ISREG(info.st_mode)) {
                continue;
            }

            // Load simulation module
            Chains s(N);
            s.SetRngSeed(r);
            s.Debug(0);
            s.SetPath(pathName);

            s.SetSamples(100);
            s.SetLength(350);

            // Set the cell dimensions
            s.CellLength(Ld);
            s.CellRadius(R);

            // Set the angle between cells
            s.CellBendingAngle(theta);

            // Let the cells relax
            s.TimeStep(dT_relax);
            s.Relax();

            // Lock the configuration
            s.CellLock();

            // Configure the phage invasion
            s.PhageInvasionStartTime(0.0);
            s.PhageInitialDensity(-1e5);

            // Set the data to export
            s.ExportColonySize();

            // Set the time step
            s.TimeStep(dT[t]);

            // Run the experiment
            s.Run(T);

        }
    }

    cout << "\rDone!                   " << endl;
	return 0;
}
